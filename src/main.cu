#include <iostream>
#include <chrono>
#include <cutf/cublas.hpp>
#include <cutf/memory.hpp>
#include <cutf/type.hpp>

enum gemm_type_t {
	FP32,
	FP16TC,
	FP16TC_FP16DATA
};

const char* get_mode_str(const gemm_type_t gemm_mode) {
	switch(gemm_mode) {
	case FP32:
		return "FP32";
	case FP16TC:
		return "FP16TC";
	case FP16TC_FP16DATA:
		return "FP16TC_FP16DATA";
	default:
		return "Unknown";
	}
	return "Unknown";
}

constexpr unsigned num_tests = 64;

void eval_gemm (
		const gemm_type_t gemm_type,
		const unsigned min_log_N,
		const unsigned max_log_N
		) {
	const auto mat_a_size = (1lu << (2 * max_log_N)) * (gemm_type == FP16TC_FP16DATA ? sizeof(half) : sizeof(float));
	const auto mat_b_size = (1lu << (2 * max_log_N)) * (gemm_type == FP16TC_FP16DATA ? sizeof(half) : sizeof(float));
	const auto mat_c_size = (1lu << (2 * max_log_N)) * sizeof(float);

	void *mat_a, *mat_b, *mat_c;
	hipMalloc(&mat_a, mat_a_size);
	hipMalloc(&mat_b, mat_b_size);
	hipMalloc(&mat_c, mat_c_size);

	auto cublas_handle_uptr = cutf::cublas::get_cublas_unique_ptr();
	hipblasGemmAlgo_t gemm_algo = HIPBLAS_GEMM_DEFAULT;
	hipblasComputeType_t compute_type;
	switch (gemm_type) {
	case FP32:
		compute_type = HIPBLAS_COMPUTE_32F;
		break;
	case FP16TC:
		compute_type = HIPBLAS_COMPUTE_32F_FAST_16F;
		break;
	case FP16TC_FP16DATA:
		compute_type = HIPBLAS_COMPUTE_32F;
		gemm_algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
		break;
	default:
		break;
	}

	for (unsigned log_N = min_log_N; log_N <= max_log_N; log_N++) {
		const auto N = 1lu << log_N;

		const float alpha = 1.0f;
		const float beta = 0.0f;

		CUTF_CHECK_ERROR(hipDeviceSynchronize());
		const auto start_clock = std::chrono::system_clock::now();

		for (unsigned t = 0; t < num_tests; t++) {
			CUTF_CHECK_ERROR(hipblasGemmEx(
						*cublas_handle_uptr.get(),
						HIPBLAS_OP_N,
						HIPBLAS_OP_N,
						N, N, N,
						&alpha,
						mat_a, (gemm_type == FP16TC_FP16DATA ? HIP_R_16F : HIP_R_32F), N,
						mat_b, (gemm_type == FP16TC_FP16DATA ? HIP_R_16F : HIP_R_32F), N,
						&beta,
						mat_c, HIP_R_32F, N,
						compute_type,
						gemm_algo
						));
		}

		CUTF_CHECK_ERROR(hipDeviceSynchronize());
		const auto end_clock = std::chrono::system_clock::now();

		const auto elapsed_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end_clock - start_clock).count() * 1e-9 / num_tests;

		std::printf("%s,%lu,%e,%e\n",
				get_mode_str(gemm_type),
				N,
				elapsed_time,
				2lu * N * N * N / elapsed_time * 1e-12
				);
		std::fflush(stdout);
	}

	hipFree(mat_a);
	hipFree(mat_b);
	hipFree(mat_c);
}

int main(int argc, char** argv) {
	if (argc < 1 + 3) {
		std::fprintf(stderr, "Usage: %s [min_log_N] [max_log_N] [mode list: FP32 FP16TC FP16TC_FP16DATA]\n", argv[0]);
		return 1;
	}

	const auto min_log_N = std::stoul(argv[1]);
	const auto max_log_N = std::stoul(argv[2]);

	std::printf("mode,N,time,throughput_in_tflops\n");
	for (unsigned i = 3; i < argc; i++) {
		const std::string mode_str = argv[i];
		gemm_type_t gemm_type = FP32;
		if (mode_str == "FP16TC") {
			gemm_type = FP16TC;
		} else if (mode_str == "FP16TC_FP16DATA") {
			gemm_type = FP16TC_FP16DATA;
		}

		eval_gemm(gemm_type, min_log_N, max_log_N);
	}
}
